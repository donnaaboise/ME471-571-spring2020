
#include <hip/hip_runtime.h>
#include <stdio.h>

__device__ int addem( int a, int b ) 
{
    return a + b;
}

__global__ void add( int a, int b, int *c ) 
{
    *c = addem( a, b );
}

int main(void) 
{
    int a,b,c;
    int *dev_c;

    /* Allocate memory on the device */
    hipMalloc( (void**)&dev_c, sizeof(int));

    a = 2;
    b = 7;
    add<<<1,1>>>(a, b, dev_c );

    /* Copy contens of dev_c back to c */
    hipMemcpy( &c, dev_c, sizeof(int), hipMemcpyDeviceToHost);

    printf( "%d + %d = %d\n", a,b,c );

    hipFree( dev_c);

}


