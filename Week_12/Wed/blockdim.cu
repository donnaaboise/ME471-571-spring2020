
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <sys/time.h>

#define P (1<<16)

__global__ void copymat_x(int m, int n, int* A, int *B) 
{    
    int idx, ix;
    int iy = threadIdx.y + blockIdx.y*blockDim.y;
    if (iy < n) 
        for(ix = 0; ix < P; ix++) {
            idx  = iy*m + ix; 
            B[idx]   = A[idx];
        }
}

__global__ void copymat_y(int m, int n, int* A, int *B) 
{    
    int ix = threadIdx.x + blockIdx.x*blockDim.x;
    int idx, iy;
    if (ix < m)
        for(iy = 0; iy < P; iy++) {
            idx  = iy*m + ix; 
            B[idx]   = A[idx];        
        }
}

double cpuSecond()
{
    struct timeval tp;
    gettimeofday(&tp,NULL);
    return (double) tp.tv_sec + (double)tp.tv_usec*1e-6;
}

int main(int argc, char** argv) 
{
    size_t m = 1 << 16;  
    size_t n = 1 << 16;  
    size_t nbytes = m*n*sizeof(int);

    printf("P = %d\n",P);

    int* A = (int*) malloc(nbytes);
    int *B = (int*) malloc(nbytes);

    memset(A,0,nbytes);

    int *dev_A, *dev_B;
    hipMalloc((void**) &dev_A, nbytes);
    hipMalloc((void**) &dev_B, nbytes);
    hipMemcpy(dev_A, A, nbytes, hipMemcpyHostToDevice);

#if 0
    /* One thread per row */
    dim3 block(1,32);  
    dim3 grid(1,(n+block.y-1)/block.y);
    double start = cpuSecond();
    copymat_x<<<grid,block>>>(m,n,dev_A, dev_B);
#else
    /* One thread per column */
    dim3 block(32,1);  
    dim3 grid((m+block.x-1)/block.x,1);
    double start = cpuSecond();
    copymat_y<<<grid,block>>>(m,n,dev_A, dev_B);
#endif
    hipDeviceSynchronize();
    double etime = cpuSecond() - start;
    printf("GPU Kernel %10.3g (s)\n",etime);

    hipFree(dev_A);
    hipFree(dev_B);
    free(A);
    free(B);

    hipDeviceReset();
}


